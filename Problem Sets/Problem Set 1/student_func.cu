#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Grean and Blue is in it.
//The 'A' stands for Alpha and is used for transparency, it will be
//ignored in this homework.

//Each channel Red, Blue, Green and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "utils.h"

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols,
                       int block_size)
{
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion
  int bx = blockIdx.x;
  int by = blockIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int pixel_x = bx * block_size + tx;
  int pixel_y = by * block_size + ty;

  if (pixel_x >= numCols || pixel_y >= numRows)
    return;
  int pixel_idx = pixel_y * numCols + pixel_x;

  uchar4 rgba = rgbaImage[pixel_idx];
  greyImage[pixel_idx] = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
}

int round_next_pow2(int x, int min_pow)
{
  int l, m, r;
  l = min_pow - 1;
  r = 31;
  while (r - l > 1) {
    m = (r + l) / 2;
    if ((1 << m) < x)
      l = m;
    else
      r = m;
  }
  return 1 << r;
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  const int block_size = 32;
  const int block_size_log2 = 5;
  const dim3 blockSize(block_size, block_size); // 1024 threads per block
  int numRowsRounded = round_next_pow2(numRows, block_size_log2);
  int numColsRounded = round_next_pow2(numCols, block_size_log2);
  const dim3 gridSize(numColsRounded / block_size, numRowsRounded / block_size);
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols, block_size);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}
